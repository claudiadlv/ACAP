
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>     
#include <ctime>       
#include <sys/time.h>  

#define IMDEP 256
#define SIZE (100*1024*1024) // 100 MB
#define THREADS_PER_BLOCK 32 // Mejor valor encontrado anteriormente

const int numRuns = 10;

double get_wall_time(){
    struct timeval time;
    if (gettimeofday(&time,NULL)){
        printf("Error en la medicion de tiempo CPU!!\n");
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * .000001;
}

void* inicializarImagen(unsigned long nBytes){
    unsigned char * img = (unsigned char*) malloc( nBytes );
    
    for(unsigned long i = 0; i<nBytes; i++){
            img[i] = rand() % IMDEP;
    }
    return img;     
}

void histogramaCPU(unsigned char* img, unsigned long nBytes, unsigned int* histo){
    double aux0 = get_wall_time();
    
    for(int i = 0; i<IMDEP; i++){
        histo[i] = 0;//Inicializacion
    }
    
    for(unsigned long i = 0; i<nBytes; i++){
        histo[img[i]]++;
    }
    
    double aux1 = get_wall_time();
    
    printf("Tiempo de CPU (s): %.4lf\n", aux1-aux0);
}

long calcularCheckSum(unsigned int* histo){
    long checkSum = 0;
    
    for(int i = 0; i<IMDEP; i++){
            checkSum += histo[i];
    }
    
    return checkSum;
}

int compararHistogramas(unsigned int* histA, unsigned int* histB){
    int valido = 1; 
    
    for(int i = 0; i<IMDEP; i++){
        if(histA[i] != histB[i]){
            printf("Error en [%d]: %u != %u\n", i, histA[i], histB[i]);
            valido = 0;
        }
    }
    return valido;
}

__global__ void kernelHistograma(unsigned char* imagen, unsigned long size, unsigned int* histo){
    
    __shared__ unsigned int temp[IMDEP];
    int focus = threadIdx.x;

    __syncthreads();

    while (focus < IMDEP){
        temp[focus] = 0;
        focus += blockDim.x;
    }
    
    unsigned long i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;
    
    while (i < size) {
        atomicAdd( &temp[imagen[i]], 1);
        i += offset;
    }
    
    __syncthreads();
    focus = threadIdx.x;
    
    while ( focus <IMDEP){
        atomicAdd( &(histo[focus]), temp[focus] );
        focus += blockDim.x;
    }
}

int main(void){
    unsigned char* imagen = (unsigned char*) inicializarImagen(SIZE);
    unsigned int histoCPU[IMDEP];
    histogramaCPU(imagen, SIZE, histoCPU);
    long chk = calcularCheckSum(histoCPU);
    printf("Check-sum CPU: %ld\n", chk);

    unsigned char *dev_imagen = 0;
    unsigned int *dev_histo = 0;
    hipMalloc( (void**) &dev_imagen, SIZE );
    hipMemcpy( dev_imagen, imagen, SIZE, hipMemcpyHostToDevice );

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliSeconds = 0.0;
    float aveGPUMS = 0.0;

    // Valores de NBLOCKS a probar
    int gridSizes[] = {512, 1024, 2048, 4096};

    for(int gs_index = 0; gs_index < 4; gs_index++) {
        int NBLOCKS = gridSizes[gs_index];

        hipMalloc( (void**) &dev_histo, IMDEP * sizeof( unsigned int) );

        for(int iter = -1; iter<numRuns; iter++){
            hipMemset( dev_histo, 0, IMDEP * sizeof( unsigned int ) );
            if(iter<0){
                kernelHistograma<<<NBLOCKS, THREADS_PER_BLOCK>>>(dev_imagen, SIZE, dev_histo);
            } else {
                hipDeviceSynchronize();
                hipEventRecord(start);
                kernelHistograma<<<NBLOCKS, THREADS_PER_BLOCK>>>(dev_imagen, SIZE, dev_histo);
                hipEventRecord(stop);
                hipEventSynchronize(stop);
                hipEventElapsedTime(&milliSeconds, start, stop);
                aveGPUMS += milliSeconds;
            }
        }
        hipFree(dev_histo);
        
        printf("Tiempo medio de ejecucion del kernel<<<%d, %d>>> sobre %u bytes [s]: %.4f\n", NBLOCKS, THREADS_PER_BLOCK, SIZE, aveGPUMS / (numRuns*1000.0));
    }

    free(imagen);
    hipFree(dev_imagen);
    
    return 0;
}
